#include "hip/hip_runtime.h"
__global__ void heston_simulate(
    float *d_paths, 
    float *d_vols, 
    float S0, float v0, float r, 
    float kappa, float theta, float sigma, float rho, 
    float T, int N, int n_paths
) {
    half h_S0 = __float2half(S0);
    half h_v0 = __float2half(v0);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_paths) return;

    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    
    float S = S0;
    float v = v0;
    d_paths[idx * (N+1)] = S0;
    d_vols[idx * (N+1)] = v0;
    
    for (int i=1; i<=N; i++) {
        float dt = T/N;
        float z1 = hiprand_normal(&state);
        float z2 = rho * z1 + sqrtf(1 - rho*rho) * hiprand_normal(&state);
        
        // Volatility process
        v = fmaxf(v + kappa*(theta - v)*dt + sigma*sqrtf(v*dt)*z1, 0.001f);
        
        // Price process
        S = S * expf((r - 0.5*v)*dt + sqrtf(v*dt)*z2);
        
        d_paths[idx*(N+1)+i] = S;
        d_vols[idx*(N+1)+i] = v;
    }
}